#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cmath>
#include <iostream>

#include "cudacheck.h"
#include "fitcuda.h"
#include "fitserial.h"

#define COARSE_FACTOR 8
#define THREADS_PER_BLOCK 1024
#define GRID_DIMENSION 80

// Patron de réduction vu en classe
__global__ void kernel_sum_coarse(double* input, double* result, int size) {
  // mémoire partagée par le warp
  // doit être allouée au lancement
  // on obtient en pratique le début de cet espace
  extern __shared__ double input_s[];

  unsigned int segment = COARSE_FACTOR * 2 * blockDim.x * blockIdx.x;
  unsigned int i = segment + threadIdx.x;
  unsigned int t = threadIdx.x;

  // Somme des éléments jusqu'à obtenir un seul bloc
  double sum_local = (i < size) ? input[i] : 0.0;
  for (unsigned int tile = 1; tile < COARSE_FACTOR * 2; tile++) {
    unsigned int src = i + tile * blockDim.x;
    if (src < size) {
      sum_local += input[src];
    }
  }

  input_s[t] = sum_local;

  // Réduction en mémoire partagée (et non en mémoire globale)
  // si blockDim = 32, stride = 16, 8, 4, 2, 1 (5 itérations)
  for (int stride = blockDim.x / 2; stride >= 1; stride >>= 1) {
    __syncthreads();
    if (t < stride) {
      input_s[t] += input_s[t + stride];
    }
  }

  // Ajouter le résultat du bloc
  if (t == 0) {
    atomicAdd(result, input_s[0]);
  }
}

__global__ void kernel_sum_coarse_one_dep(double* input, double* mean ,double* result, int size) {
  // mémoire partagée par le warp
  // doit être allouée au lancement
  // on obtient en pratique le début de cet espace
  extern __shared__ double input_s[];

  unsigned int segment = COARSE_FACTOR * 2 * blockDim.x * blockIdx.x;
  unsigned int i = segment + threadIdx.x;
  unsigned int t = threadIdx.x;

  // double sum_local = (i < size) ? input[i] - *mean : 0.0;
  double sum_local = 0.0;

  for (unsigned int tile = 0; tile < COARSE_FACTOR * 2; tile++) {
    unsigned int src = i + tile * blockDim.x;
    if (src < size) {
      // sum_local = sum_local + (input[src] - *mean) * (input[src] - *mean);
      double diff = input[src] - *mean;
      sum_local += diff * diff;
    }
  }

  input_s[t] = sum_local;

  // Réduction en mémoire partagée (et non en mémoire globale)
  // si blockDim = 32, stride = 16, 8, 4, 2, 1 (5 itérations)
  for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
    __syncthreads();
    if (t < stride) {
      // input_s[t] = input_s[t] + (input_s[t + stride] - *mean) * (input_s[t + stride] - *mean);
      input_s[t] += input_s[t + stride];
    }
  }

  // Ajouter le résultat du bloc
  if (t == 0) {
    atomicAdd(result, input_s[0]);
  }
}

__global__ void kernel_sum_coarse_two_dep(double* xArray, double* xmean, double* yArray, double* ymean, double* result, int size) {
  // mémoire partagée par le warp
  // doit être allouée au lancement
  // on obtient en pratique le début de cet espace
  extern __shared__ double input_s[];

  unsigned int segment = COARSE_FACTOR * 2 * blockDim.x * blockIdx.x;
  unsigned int i = segment + threadIdx.x;
  unsigned int t = threadIdx.x;

  double sum_local = 0.0;

  for (unsigned int tile = 0; tile < COARSE_FACTOR * 2; tile++) {
    unsigned int src = i + tile * blockDim.x;
    if (src < size) {
      double diff_x = xArray[src] - *xmean;
      double diff_y = yArray[src] - *ymean;
      sum_local += diff_x * diff_y;
    }
  }

  input_s[t] = sum_local;

  // Réduction en mémoire partagée (et non en mémoire globale)
  // si blockDim = 32, stride = 16, 8, 4, 2, 1 (5 itérations)
  for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
    __syncthreads();
    if (t < stride) {
      input_s[t] += input_s[t + stride];
    }
  }

  // Ajouter le résultat du bloc
  if (t == 0) {
    atomicAdd(result, input_s[0]);
  }
}

FitCuda::FitCuda() {
  // test pour vérifier que nous avons bel et bien un périphérique
  int deviceId;
  cudaCheck(hipGetDevice(&deviceId));
}

FitCuda::~FitCuda() {}

void FitCuda::fit(double* xarray, double* yarray, int size, FitResult& res) {
  double a = 0;
  double b = 0;
  double r = 0;
  double sx = 0.0;
  double sy = 0.0;
  double ss = size;
  double xmean = 0.0;
  double ymean = 0.0;
  double ssxym = 0.0;
  double ssxm = 0.0;
  double ssym = 0.0;

/******************************************************************/
/* sum(x) and sum(y)
/******************************************************************/

  // double result;
  double* xarray_d;
  double* yarray_d;
  double* xmean_d;
  double* ymean_d;
  double* result_d;

  hipMalloc(&xarray_d, size * sizeof(double));
  hipMalloc(&yarray_d, size * sizeof(double));
  hipMalloc(&xmean_d, sizeof(double));
  hipMalloc(&ymean_d, sizeof(double));

  hipMalloc(&result_d, sizeof(double));

  hipMemcpy(xarray_d, xarray, size * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(yarray_d, yarray, size * sizeof(double), hipMemcpyHostToDevice);
  hipMemset(result_d, 0, sizeof(double));

  // Pour la fraction de GPU V100
  int blockDim = THREADS_PER_BLOCK;
  int gridDim = (size + blockDim * COARSE_FACTOR * 2 - 1) / (blockDim * COARSE_FACTOR * 2);
  // int gridDim = GRID_DIMENSION;
  int sharedSize = blockDim * sizeof(double);  // taille du tableau extern __shared__ double input_s[]
  kernel_sum_coarse<<<gridDim, blockDim, sharedSize>>>(xarray_d, result_d, size);
  cudaCheck(hipDeviceSynchronize());
  hipMemcpy(&sx, result_d, sizeof(double), hipMemcpyDeviceToHost);
  hipMemset(result_d, 0, sizeof(double));
  kernel_sum_coarse<<<gridDim, blockDim, sharedSize>>>(yarray_d, result_d, size);
  cudaCheck(hipDeviceSynchronize());
  hipMemcpy(&sy, result_d, sizeof(double), hipMemcpyDeviceToHost);
  hipMemset(result_d, 0, sizeof(double));


/******************************************************************/
/* sxxm, ssym, ssxym
/******************************************************************/

  xmean = sx / size;
  ymean = sy / size;

  hipMemcpy(xmean_d, &xmean, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(ymean_d, &ymean, sizeof(double), hipMemcpyHostToDevice);

  // ssxm = reduction_one_dep(xarray, xmean, size);
  kernel_sum_coarse_one_dep<<<gridDim, blockDim, sharedSize>>>(xarray_d, xmean_d, result_d, size);
  cudaCheck(hipDeviceSynchronize());
  hipMemcpy(&ssxm, result_d, sizeof(double), hipMemcpyDeviceToHost);
  hipMemset(result_d, 0, sizeof(double));

  // ssym = reduction_one_dep(yarray, ymean, size);
  kernel_sum_coarse_one_dep<<<gridDim, blockDim, sharedSize>>>(yarray_d, ymean_d, result_d, size);
  cudaCheck(hipDeviceSynchronize());
  hipMemcpy(&ssym, result_d, sizeof(double), hipMemcpyDeviceToHost);
  hipMemset(result_d, 0, sizeof(double));

  // ssxym = reduction_two_dep(xarray, xmean, yarray, ymean, size);
  kernel_sum_coarse_two_dep<<<gridDim, blockDim, 2*sharedSize>>>(xarray_d, xmean_d, yarray_d, ymean_d, result_d, size);
  cudaCheck(hipDeviceSynchronize());
  hipMemcpy(&ssxym, result_d, sizeof(double), hipMemcpyDeviceToHost);
  hipMemset(result_d, 0, sizeof(double));

  std::cout << "\nssxm: " << ssxm << std::endl;
  std::cout << "ssym: " << ssym << std::endl;
  std::cout << "ssxym: " << ssxym << std::endl;

  b = ssxym / ssxm;
  a = (sy - sx * b) / ss;

  ssxm = ssxm / size;
  ssym = ssym / size;
  ssxym = ssxym / size;

  // Coefficient R
  if ((ssxm != 0.0) && (ssym != 0.0)) {
    r = ssxym / sqrt(ssxm * ssym);
    if (r > 1.0) {
      r = 1.0;
    } else if (r < -1.0) {
      r = -1.0;
    }
  }

  res.a = a;
  res.b = b;
  res.r = r;
  res.xmean = xmean;
  res.ymean = ymean;

  hipFree(xarray_d);
  hipFree(yarray_d);
  hipFree(xmean_d);
  hipFree(ymean_d);
  hipFree(result_d);
}

// Only kept to pass the tests, not used in the fit function
double FitCuda::reduction(double* array, int size) {
  double result;
  double* array_d;
  double* result_d;
  hipMalloc(&array_d, size * sizeof(double));
  hipMalloc(&result_d, sizeof(double));

  hipMemcpy(array_d, array, size * sizeof(double), hipMemcpyHostToDevice);
  hipMemset(result_d, 0, sizeof(double));

  // Pour la fraction de GPU V100
  int blockDim = 1024;
  int gridDim = 80;
  int sharedSize = blockDim * sizeof(double);  // taille du tableau extern __shared__ double input_s[]
  kernel_sum_coarse<<<gridDim, blockDim, sharedSize>>>(array_d, result_d, size);
  cudaCheck(hipDeviceSynchronize());

  hipMemcpy(&result, result_d, sizeof(double), hipMemcpyDeviceToHost);
  return result;
}


// double FitCuda::reduction_one_dep(double* xArray, double xmean, int size) {
//   double result;
//   double* xArray_d;
//   double* xmean_d;
//   double* result_d;
//   hipMalloc(&xArray_d, size * sizeof(double));
//   hipMalloc(&xmean_d, sizeof(double));
//   hipMalloc(&result_d, sizeof(double));

//   hipMemcpy(xArray_d, xArray, size * sizeof(double), hipMemcpyHostToDevice);
//   hipMemcpy(xmean_d, &xmean, sizeof(double), hipMemcpyHostToDevice);
//   hipMemset(result_d, 0, sizeof(double));

//   // Pour la fraction de GPU V100
//   int blockDim = 1024;
//   int gridDim = 80;
//   int sharedSize = blockDim * sizeof(double);  // taille du tableau extern __shared__ double input_s[]
//   kernel_sum_coarse_one_dep<<<gridDim, blockDim, sharedSize>>>(xArray_d, xmean_d, result_d, size);
//   cudaCheck(hipDeviceSynchronize());

//   hipMemcpy(&result, result_d, sizeof(double), hipMemcpyDeviceToHost);
//   return result;
// }

