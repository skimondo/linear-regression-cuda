#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cmath>
#include <iostream>

#include "cudacheck.h"
#include "fitcuda.h"
#include "fitserial.h"

#define COARSE_FACTOR 8
#define THREADS_PER_BLOCK 1024
#define GRID_DIMENSION 80


// Patron de réduction vu en classe
__global__ void kernel_sum_coarse(double* input, double* result, int size) {
  // mémoire partagée par le warp
  // doit être allouée au lancement
  // on obtient en pratique le début de cet espace
  extern __shared__ double input_s[];

  unsigned int segment = COARSE_FACTOR * 2 * blockDim.x * blockIdx.x;
  unsigned int i = segment + threadIdx.x;
  unsigned int t = threadIdx.x;

  // Somme des éléments jusqu'à obtenir un seul bloc
  double sum_local = (i < size) ? input[i] : 0.0;
  for (unsigned int tile = 1; tile < COARSE_FACTOR * 2; tile++) {
    unsigned int src = i + tile * blockDim.x;
    if (src < size) {
      sum_local += input[src];
    }
  }

  input_s[t] = sum_local;

  // Réduction en mémoire partagée (et non en mémoire globale)
  // si blockDim = 32, stride = 16, 8, 4, 2, 1 (5 itérations)
  for (int stride = blockDim.x / 2; stride >= 1; stride >>= 1) {
    __syncthreads();
    if (t < stride) {
      input_s[t] += input_s[t + stride];
    }
  }

  // Ajouter le résultat du bloc
  if (t == 0) {
    atomicAdd(result, input_s[0]);
  }
}

__global__ void kernel_sum_coarse_one_dep(double* input, double* mean ,double* result, int size) {
  // mémoire partagée par le warp
  // doit être allouée au lancement
  // on obtient en pratique le début de cet espace
  extern __shared__ double input_s[];

  unsigned int segment = COARSE_FACTOR * 2 * blockDim.x * blockIdx.x;
  unsigned int i = segment + threadIdx.x;
  unsigned int t = threadIdx.x;

  // double sum_local = (i < size) ? input[i] - *mean : 0.0;
  double sum_local = 0.0;

  for (unsigned int tile = 0; tile < COARSE_FACTOR * 2; tile++) {
    unsigned int src = i + tile * blockDim.x;
    if (src < size) {
      // sum_local = sum_local + (input[src] - *mean) * (input[src] - *mean);
      double diff = input[src] - *mean;
      sum_local += diff * diff;
    }
  }

  input_s[t] = sum_local;

  // Réduction en mémoire partagée (et non en mémoire globale)
  // si blockDim = 32, stride = 16, 8, 4, 2, 1 (5 itérations)
  for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
    __syncthreads();
    if (t < stride) {
      // input_s[t] = input_s[t] + (input_s[t + stride] - *mean) * (input_s[t + stride] - *mean);
      input_s[t] += input_s[t + stride];
    }
  }

  // Ajouter le résultat du bloc
  if (t == 0) {
    atomicAdd(result, input_s[0]);
  }
}

__global__ void kernel_sum_coarse_two_dep(double* xArray, double* xmean, double* yArray, double* ymean, double* result, int size) {
  // mémoire partagée par le warp
  // doit être allouée au lancement
  // on obtient en pratique le début de cet espace
  extern __shared__ double input_s[];

  unsigned int segment = COARSE_FACTOR * 2 * blockDim.x * blockIdx.x;
  unsigned int i = segment + threadIdx.x;
  unsigned int t = threadIdx.x;

  double sum_local = 0.0;

  for (unsigned int tile = 0; tile < COARSE_FACTOR * 2; tile++) {
    unsigned int src = i + tile * blockDim.x;
    if (src < size) {
      double diff_x = xArray[src] - *xmean;
      double diff_y = yArray[src] - *ymean;
      sum_local += diff_x * diff_y;
    }
  }

  input_s[t] = sum_local;

  // Réduction en mémoire partagée (et non en mémoire globale)
  // si blockDim = 32, stride = 16, 8, 4, 2, 1 (5 itérations)
  for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
    __syncthreads();
    if (t < stride) {
      input_s[t] += input_s[t + stride];
    }
  }

  // Ajouter le résultat du bloc
  if (t == 0) {
    atomicAdd(result, input_s[0]);
  }
}

FitCuda::FitCuda() {
  // test pour vérifier que nous avons bel et bien un périphérique
  int deviceId;
  cudaCheck(hipGetDevice(&deviceId));
}

FitCuda::~FitCuda() {}

void FitCuda::fit(double* xarray, double* yarray, int size, FitResult& res) {
  double a = 0;
  double b = 0;
  double r = 0;
  double sx = 0.0;
  double sy = 0.0;
  double ss = size;
  double xmean = 0.0;
  double ymean = 0.0;
  double ssxym = 0.0;
  double ssxm = 0.0;
  double ssym = 0.0;


  // SIMPLE REDUCTION PART 

  sx = reduction(xarray, size);
  sy = reduction(yarray, size);

  // END OF SIMPLE REDUCTION PART

  xmean = sx / size;
  ymean = sy / size;

  // REDUCTION WITH ONE DEPENDENCY PART

  ssxm = reduction_one_dep(xarray, xmean, size);
  ssym = reduction_one_dep(yarray, ymean, size);

  // END OF REDUCTION WITH ONE DEPENDENCY PART

  // REDUCTION WITH TWO DEPENDENCIES PART

  ssxym = reduction_two_dep(xarray, xmean, yarray, ymean, size);

  // END OF REDUCTION WITH TWO DEPENDENCIES PART

  b = ssxym / ssxm;
  a = (sy - sx * b) / ss;

  ssxm = ssxm / size;
  ssym = ssym / size;
  ssxym = ssxym / size;

  // Coefficient R
  if ((ssxm != 0.0) && (ssym != 0.0)) {
    r = ssxym / sqrt(ssxm * ssym);
    if (r > 1.0) {
      r = 1.0;
    } else if (r < -1.0) {
      r = -1.0;
    }
  }

  res.a = a;
  res.b = b;
  res.r = r;
  res.xmean = xmean;
  res.ymean = ymean;
}


double FitCuda::reduction(double* array, int size) {
  double result;
  double* array_d;
  double* result_d;
  hipMalloc(&array_d, size * sizeof(double));
  hipMalloc(&result_d, sizeof(double));

  hipMemcpy(array_d, array, size * sizeof(double), hipMemcpyHostToDevice);
  hipMemset(result_d, 0, sizeof(double));

  // Pour la fraction de GPU V100
  int blockDim = 1024;
  // int gridDim = 80;
  int gridDim = (size + blockDim * COARSE_FACTOR * 2 - 1) / (blockDim * COARSE_FACTOR * 2);
  int sharedSize = blockDim * sizeof(double);  // taille du tableau extern __shared__ double input_s[]
  kernel_sum_coarse<<<gridDim, blockDim, sharedSize>>>(array_d, result_d, size);
  cudaCheck(hipDeviceSynchronize());

  hipMemcpy(&result, result_d, sizeof(double), hipMemcpyDeviceToHost);

  hipFree(array_d);
  hipFree(result_d);

  return result;
}


double FitCuda::reduction_one_dep(double* xArray, double xmean, int size) {
  double result;
  double* xArray_d;
  double* xmean_d;
  double* result_d;
  hipMalloc(&xArray_d, size * sizeof(double));
  hipMalloc(&xmean_d, sizeof(double));
  hipMalloc(&result_d, sizeof(double));

  hipMemcpy(xArray_d, xArray, size * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(xmean_d, &xmean, sizeof(double), hipMemcpyHostToDevice);
  hipMemset(result_d, 0, sizeof(double));

  // Pour la fraction de GPU V100
  int blockDim = 1024;
  int gridDim = 80;
  int sharedSize = blockDim * sizeof(double);  // taille du tableau extern __shared__ double input_s[]
  kernel_sum_coarse_one_dep<<<gridDim, blockDim, sharedSize>>>(xArray_d, xmean_d, result_d, size);
  cudaCheck(hipDeviceSynchronize());

  hipMemcpy(&result, result_d, sizeof(double), hipMemcpyDeviceToHost);

  hipFree(xArray_d);
  hipFree(xmean_d);
  hipFree(result_d);

  return result;
}

double FitCuda::reduction_two_dep(double* xArray, double xmean, double* yArray, double ymean, int size) {
  double result;
  double* xArray_d;
  double* xmean_d;
  double* yArray_d;
  double* ymean_d;
  double* result_d;
  hipMalloc(&xArray_d, size * sizeof(double));
  hipMalloc(&xmean_d, sizeof(double));
  hipMalloc(&yArray_d, size * sizeof(double));
  hipMalloc(&ymean_d, sizeof(double));
  hipMalloc(&result_d, sizeof(double));

  hipMemcpy(xArray_d, xArray, size * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(xmean_d, &xmean, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(yArray_d, yArray, size * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(ymean_d, &ymean, sizeof(double), hipMemcpyHostToDevice);
  hipMemset(result_d, 0, sizeof(double));

  // Pour la fraction de GPU V100
  int blockDim = 1024;
  int gridDim = 80;
  int sharedSize = blockDim * sizeof(double);  // taille du tableau extern __shared__ double input_s[]
  kernel_sum_coarse_two_dep<<<gridDim, blockDim, sharedSize>>>(xArray_d, xmean_d, yArray_d, ymean_d, result_d, size);
  cudaCheck(hipDeviceSynchronize());

  hipMemcpy(&result, result_d, sizeof(double), hipMemcpyDeviceToHost);

  hipFree(xArray_d);
  hipFree(xmean_d);
  hipFree(yArray_d);
  hipFree(ymean_d);
  hipFree(result_d);

  return result;
}